#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <cmath>

#define COUNT_UNIT8_T_HASH 16

#define MD5_CHUNKS_BYTE 512/8
#define MD5_TEXT_LEN 448/8

//#define DEBUG

#define LEFTROTATE(x, c) (((x) << (c)) | ((x) >> (32 - (c))))

__host__ __device__
void to_bytes(uint32_t val, uint8_t *bytes) {
    bytes[0] = (uint8_t) val;
    bytes[1] = (uint8_t) (val >> 8);
    bytes[2] = (uint8_t) (val >> 16);
    bytes[3] = (uint8_t) (val >> 24);
}

__host__ __device__
uint32_t to_int32(const uint8_t *bytes) {
    return (uint32_t) bytes[0] | ((uint32_t) bytes[1] << 8)
           | ((uint32_t) bytes[2] << 16) | ((uint32_t) bytes[3] << 24);
}

__host__ __device__
void md5(const uint8_t *initial_msg, size_t initial_len, uint8_t *digest) {

    // Constants are the integer part of the sines of integers (in radians) * 2^32.
    const uint32_t k[64] = {0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee,
                            0xf57c0faf, 0x4787c62a, 0xa8304613, 0xfd469501, 0x698098d8, 0x8b44f7af,
                            0xffff5bb1, 0x895cd7be, 0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821,
                            0xf61e2562, 0xc040b340, 0x265e5a51, 0xe9b6c7aa, 0xd62f105d, 0x02441453,
                            0xd8a1e681, 0xe7d3fbc8, 0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed,
                            0xa9e3e905, 0xfcefa3f8, 0x676f02d9, 0x8d2a4c8a, 0xfffa3942, 0x8771f681,
                            0x6d9d6122, 0xfde5380c, 0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70,
                            0x289b7ec6, 0xeaa127fa, 0xd4ef3085, 0x04881d05, 0xd9d4d039, 0xe6db99e5,
                            0x1fa27cf8, 0xc4ac5665, 0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039,
                            0x655b59c3, 0x8f0ccc92, 0xffeff47d, 0x85845dd1, 0x6fa87e4f, 0xfe2ce6e0,
                            0xa3014314, 0x4e0811a1, 0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391};

    // r specifies the per-round shift amounts
    const uint32_t r[] = {7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17, 22, 7, 12, 17,
                          22, 5, 9, 14, 20, 5, 9, 14, 20, 5, 9, 14, 20, 5, 9, 14, 20, 4, 11, 16,
                          23, 4, 11, 16, 23, 4, 11, 16, 23, 4, 11, 16, 23, 6, 10, 15, 21, 6, 10,
                          15, 21, 6, 10, 15, 21, 6, 10, 15, 21};

    // These vars will contain the hash
    uint32_t h0, h1, h2, h3;

    // Message (to prepare)
    uint8_t *msg = NULL;

    size_t new_len, offset;
    uint32_t w[16];
    uint32_t a, b, c, d, i, f, g, temp;

    // Initialize variables - simple count in nibbles:
    h0 = 0x67452301;
    h1 = 0xefcdab89;
    h2 = 0x98badcfe;
    h3 = 0x10325476;

    //Pre-processing:
    //append "1" bit to message
    //append "0" bits until message length in bits ≡ 448 (mod 512)
    //append length mod (2^64) to message

    for (new_len = initial_len + 1; new_len % (MD5_CHUNKS_BYTE) != MD5_TEXT_LEN; new_len++);

    msg = (uint8_t *) malloc(new_len + 8);
    memcpy(msg, initial_msg, initial_len);
    msg[initial_len] = 0x80; // append the "1" bit; most significant bit is "first"

    for (offset = initial_len + 1; offset < new_len; offset++)
        msg[offset] = 0; // append "0" bits

    // append the len in bits at the end of the buffer.
    to_bytes(initial_len * 8, msg + new_len);

    // initial_len>>29 == initial_len*8>>32, but avoids overflow.
    to_bytes(initial_len >> 29, msg + new_len + 4);

    // Process the message in successive 512-bit chunks:
    //for each 512-bit chunk of message:
    for (offset = 0; offset < new_len; offset += (MD5_CHUNKS_BYTE)) {

        // break chunk into sixteen 32-bit words w[j], 0 ≤ j ≤ 15
        for (i = 0; i < COUNT_UNIT8_T_HASH; i++)
            w[i] = to_int32(msg + offset + i * 4);

        // Initialize hash value for this chunk:
        a = h0;
        b = h1;
        c = h2;
        d = h3;

        // Main loop:
        for (i = 0; i < 64; i++) {

            if (i < 16) {
                f = (b & c) | ((~b) & d);
                g = i;
            } else if (i < 32) {
                f = (d & b) | ((~d) & c);
                g = (5 * i + 1) % 16;
            } else if (i < 48) {
                f = b ^ c ^ d;
                g = (3 * i + 5) % 16;
            } else {
                f = c ^ (b | (~d));
                g = (7 * i) % 16;
            }

            temp = d;
            d = c;
            c = b;
            b = b + LEFTROTATE((a + f + k[i] + w[g]), r[i]);
            a = temp;

        }

        // Add this chunk's hash to result so far:
        h0 += a;
        h1 += b;
        h2 += c;
        h3 += d;

    }

    // cleanup
    free(msg);

    //var char digest[16] := h0 append h1 append h2 append h3 //(Output is in little-endian)
    to_bytes(h0, digest);
    to_bytes(h1, digest + 4);
    to_bytes(h2, digest + 8);
    to_bytes(h3, digest + 12);
}

__host__ __device__
int my_strlen(char *text) {
    int len = 0;
    int i = 0;
    while (text[i++] != '\0') {
        len++;
    }
    return len;
}

__host__ __device__
void hash_md5(char *input, uint8_t *result) {
    md5((uint8_t *) input, (size_t) my_strlen(input), result);
}


__global__ void kernel_mult(char *words, const int height, const int width, uint8_t *cHashWords) {

    int l = blockDim.x * blockIdx.x + threadIdx.x;

    if (l >= height) {
        return;
    }

    char *word = new char[width + 1];

    int j;
    for (j = 0; j < width; j++) {
        word[j] = words[width * l + j];
    }
    word[j] = '\0';

    hash_md5(word, &cHashWords[COUNT_UNIT8_T_HASH * l]);

#ifdef DEBUG
    printf("debug: hashing word: %s\n", word);
#endif

    delete[] word;
}


void run_mult(char *words, const int height, const int width, uint8_t *hashed_words) {

    hipError_t cerr;
    int threads = 128;
    int length = height * width;
    int blocks = (length + threads - 1) / threads;

    char *cWords;
    cerr = hipMalloc(&cWords, length * sizeof(char));
    if (cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(cerr));

    uint8_t *cHashWords;
    cerr = hipMalloc(&cHashWords, height * COUNT_UNIT8_T_HASH * sizeof(uint8_t));
    if (cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(cerr));


    cerr = hipMemcpy(cWords, words, length * sizeof(char), hipMemcpyHostToDevice);
    if (cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(cerr));

    kernel_mult<<< blocks, threads >>> (cWords, height, width, cHashWords);

    if ((cerr = hipGetLastError()) != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(cerr));

    cerr = hipMemcpy(hashed_words, cHashWords, height * COUNT_UNIT8_T_HASH * sizeof(uint8_t), hipMemcpyDeviceToHost);
    if (cerr != hipSuccess)
        printf("CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString(cerr));

    hipFree(cWords);
    hipFree(cHashWords);
}
